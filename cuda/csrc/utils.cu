#include "hip/hip_runtime.h"

template <typename T>
__global__ void
copy_out_indices_kernel(const T *A, const int *shape_A, const int *stride_A,
                        const int **indices, const int *shape_out, T *C,
                        int ndim_A, int totalSize) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= totalSize)
    return;

  int coords_out[MAX_DIMS];
  unravel_index(idx, shape_out, ndim_A, coords_out);

  int coords_A[MAX_DIMS];
  for (int d = 0; d < ndim_A; d++) {
    coords_A[d] = indices[d][coords_out[d]];
  }

  int flat_A = flattenIndex(ndim_A, coords_A, stride_A);
  C[idx] = A[flat_A];
}

template <typename T>
void copy_out_indices(const T *d_A, const int *shape_A, const int *stride_A,
                      T *d_C, const int **indices, const int *shape_C,
                      int ndim_A
                      // ,int ndim_C
) {
  int totalSize = _size(shape_A, ndim_A);

  int blockSize = BLOCK_SIZE;
  int gridSize = (totalSize + blockSize - 1) / blockSize;

  int *d_shape_A;
  shapeToDevice(shape_A, &d_shape_A, ndim_A);
  int *d_stride_A;
  shapeToDevice(stride_A, &d_stride_A, ndim_A);

  int *d_shape_C;
  shapeToDevice(shape_C, &d_shape_C, ndim_A);

  const int **d_indices;
  hipMalloc(&d_indices, ndim_A * sizeof(int *));
  hipMemcpy(d_indices, indices, ndim_A * sizeof(int *),
             hipMemcpyHostToDevice);

  copy_out_indices_kernel<<<gridSize, blockSize>>>(
      d_A, d_shape_A, d_stride_A, d_indices, d_shape_C, d_C, ndim_A, totalSize);
}
#define COPY_OUT_INDICES(T)                                                    \
  extern "C" void copy_out_indices_##T(                                        \
      const T *d_A, const int *shape_A, const int *stride_A, T *d_C,           \
      const int **indices, const int *shape_C, int ndim_A) {                   \
    copy_out_indices(d_A, shape_A, stride_A, d_C, indices, shape_C, ndim_A);   \
  }

template <typename I, typename O>
__global__ void copy_out_kernel(const I *A, const int *shape_A,
                                const int *stride_A, O *C, int ndim_A,
                                int totalSize) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= totalSize)
    return;

  int coords_A[MAX_DIMS];
  unravel_index(idx, shape_A, ndim_A, coords_A);
  int flat_A = flattenIndex(ndim_A, coords_A, stride_A);

  C[idx] = (O)A[flat_A];
}

template <typename I, typename O>
void copy_out(const I *d_A, const int *shape_A, const int *stride_A, O *d_C,
              int ndim_A) {
  int totalSize = _size(shape_A, ndim_A);

  int blockSize = BLOCK_SIZE;
  int gridSize = (totalSize + blockSize - 1) / blockSize;

  int *d_shape_A;
  shapeToDevice(shape_A, &d_shape_A, ndim_A);
  int *d_stride_A;
  shapeToDevice(stride_A, &d_stride_A, ndim_A);

  copy_out_kernel<<<gridSize, blockSize>>>(d_A, d_shape_A, d_stride_A, d_C,
                                           ndim_A, totalSize);
}
#define COPY_OUT(I, O)                                                         \
  extern "C" void copy_out_##I##_##O(const I *d_A, const int *shape_A,         \
                                     const int *stride_A, O *d_C,              \
                                     int ndim_A) {                             \
    copy_out(d_A, shape_A, stride_A, d_C, ndim_A);                             \
  }

COPY_OUT_INDICES(int32)
COPY_OUT_INDICES(int64)
COPY_OUT_INDICES(float32)
COPY_OUT_INDICES(float64)
